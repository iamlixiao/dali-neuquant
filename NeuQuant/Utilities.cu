#include "hip/hip_runtime.h"
/*
 * Utilities.cpp
 *
 *  Created on: Oct 1, 2011
 *      Author: Dave
 */

#include "Utilities.h"

const Utilities::NodeDistance1D Utilities::nodeDistance1d =
    Utilities::NodeDistance1D();

//------------------------------------------------------------------------------
// Name:        NodeDistance1D::operator()
//------------------------------------------------------------------------------
float Utilities::NodeDistance1D::operator ()(const unsigned int idx1,
    const unsigned int idx2)
{
  return static_cast<float>(idx1 < idx2 ? idx2 - idx1 : idx1 - idx2);
}

//------------------------------------------------------------------------------
// Name:        NodeDistance2D::NodeDistance2D()
//------------------------------------------------------------------------------
Utilities::NodeDistance2D::NodeDistance2D(const unsigned int cols) : cols(cols)
{ }

//------------------------------------------------------------------------------
// Name:        NodeDistance2D::operator()
//------------------------------------------------------------------------------
float Utilities::NodeDistance2D::operator()(const unsigned int idx1,
    const unsigned int idx2)
{
  const unsigned int row1 = idx1 / cols, col1 = idx1 % cols;
  const unsigned int row2 = idx2 / cols, col2 = idx2 % cols;

  return sqrt(static_cast<float>((row1 - row2) * (row1 - row2) +
      (col1 - col2) * (col1 - col2)));
}

//------------------------------------------------------------------------------
// Name:        NodeDistance3D::NodeDistance3D()
//------------------------------------------------------------------------------
Utilities::NodeDistance3D::NodeDistance3D(const unsigned int cols,
    const unsigned int sliceSize) : cols(cols), sliceSize(sliceSize)
{ }

//------------------------------------------------------------------------------
// Name:        NodeDistance3D::operator()
//------------------------------------------------------------------------------
float Utilities::NodeDistance3D::operator()(const unsigned int idx1,
    const unsigned int idx2)
{
  const unsigned int slice1 = idx1 / sliceSize,
      idx1InSlice = idx1 % sliceSize, row1 = idx1InSlice / cols,
      col1 = idx1InSlice % cols;
  const unsigned int slice2 = idx2 / sliceSize,
      idx2InSlice = idx2 % sliceSize, row2 = idx2InSlice / cols,
      col2 = idx2InSlice % cols;
  return sqrt(static_cast<float>((slice1 - slice2) * (slice1 - slice2) +
      (row1 - row2) * (row1 - row2) +
      (col1 - col2) * (col1 - col2)));
}

