#include "hip/hip_runtime.h"
/*
 * Kohonen.cu
 *
 *  Created on: Oct 29, 2011
 *      Author: David Bottisti
 */

#include "Kohonen.h"
#include <stdexcept>
#include <cstdlib>
#include <cmath>
#include <hip/hip_runtime.h>

const unsigned int Kohonen::numInputDimensions = 3;
const unsigned int Kohonen::networkSize = 256;

// four primes near 500 - assume no image has a length so large
// that it is divisible by all four primes
const unsigned short Kohonen::primes[4] = {499, 491, 487, 503};

// Device Code
__shared__ float dist[Kohonen::networkSize];
__shared__ unsigned char bestIdx[Kohonen::networkSize];
__device__ float networkRed[Kohonen::networkSize];
__device__ float networkGreen[Kohonen::networkSize];
__device__ float networkBlue[Kohonen::networkSize];

__global__ void initializeNetwork(void)
{
  const unsigned int idx = threadIdx.x;
  networkRed[idx] = idx;
  networkGreen[idx] = idx;
  networkBlue[idx] = idx;
}

__global__ void trainOnPoint(const float * const points,
    const unsigned int step, const unsigned int pointsPerIteration)
{
  const unsigned int numInputPoints = gridDim.x * gridDim.y;
  const unsigned int tempPointIdx = blockIdx.y * gridDim.x + blockIdx.x;
  const unsigned int pointIdx = (tempPointIdx * step) % numInputPoints;
  const unsigned int networkIdx = threadIdx.x;
  const unsigned int iteration = tempPointIdx / pointsPerIteration;
  const float alpha = exp(-0.03 * iteration);
  const unsigned int radius = static_cast<unsigned int>(
      32.0 * exp(-0.0325 * iteration));

  __shared__ Kohonen::Color point;
  if (networkIdx == 0)
  {
    point.red = points[pointIdx];
    point.green = points[pointIdx + numInputPoints];
    point.blue = points[pointIdx + 2 * numInputPoints];
  }
  __syncthreads();

  // Compute the distance between the training point and the network
  dist[networkIdx] = fabs(point.red - networkRed[networkIdx]) +
      fabs(point.green - networkGreen[networkIdx]) +
      fabs(point.blue - networkBlue[networkIdx]);
  bestIdx[networkIdx] = networkIdx;

  // Wait for all the threads to compute their distance
  __syncthreads();

  // Reduce the distances
  for (unsigned int s = blockDim.x >> 1; s > 0; s >>= 1)
  {
    if (networkIdx < s && dist[networkIdx + s] < dist[networkIdx])
    {
      dist[networkIdx] = dist[networkIdx + s];
      bestIdx[networkIdx] = bestIdx[networkIdx + s];
    }

    __syncthreads();
  }
  const unsigned int winner = bestIdx[0];

  // Update the node based on the contents of the winning node, if it is within
  // the radius.
  const unsigned int dist = (winner < networkIdx ?
      networkIdx - winner : winner - networkIdx);
  if (dist < radius)
  {
    // 1) Compute alpha * rho based upon distance to winning index
    const float distOverRadius = static_cast<float>(dist) / radius;
    const float alphaRho = alpha * (1 - distOverRadius * distOverRadius);
    const float oneMinusAlphaRho = 1 - alphaRho;

    // 2) Compute the new node
    networkRed[networkIdx] = alphaRho * point.red +
        oneMinusAlphaRho * networkRed[networkIdx];
    networkGreen[networkIdx] = alphaRho * point.green +
        oneMinusAlphaRho * networkGreen[networkIdx];
    networkBlue[networkIdx] = alphaRho * point.blue +
        oneMinusAlphaRho * networkBlue[networkIdx];
  }
}

__global__ void mapPoints(float * const points)
{
  const unsigned int numInputPoints = gridDim.x * gridDim.y;
  const unsigned int pointIdx = blockIdx.y * gridDim.x + blockIdx.x;
  const unsigned int networkIdx = threadIdx.x;

  __shared__ Kohonen::Color point;
  if (networkIdx == 0)
  {
    point.red = points[pointIdx];
    point.green = points[pointIdx + numInputPoints];
    point.blue = points[pointIdx + 2 * numInputPoints];
  }
  __syncthreads();

  // Compute the distance between the image point and the network
  dist[networkIdx] = fabs(point.red - networkRed[networkIdx]) +
      fabs(point.green - networkGreen[networkIdx]) +
      fabs(point.blue - networkBlue[networkIdx]);
  bestIdx[networkIdx] = networkIdx;

  // Wait for all the threads in a block to compute their distance
  __syncthreads();

  // Reduce the distances by block
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
  {
    if (networkIdx < s && dist[networkIdx + s] < dist[networkIdx])
    {
      dist[networkIdx] = dist[networkIdx + s];
      bestIdx[networkIdx] = bestIdx[networkIdx + s];
    }

    __syncthreads();
  }

  // Copy the winning node to the output
  if (networkIdx == 0)
  {
    const unsigned int winningIdx = bestIdx[0];
    points[pointIdx] = networkRed[winningIdx];
    points[pointIdx + numInputPoints] = networkGreen[winningIdx];
    points[pointIdx + 2 * numInputPoints] = networkBlue[winningIdx];
  }
}

Kohonen::Kohonen(void)
{
  // Initialize the network
  initializeNetwork<<<1, networkSize>>>();
}

Kohonen::~Kohonen()
{
}

void Kohonen::train(const unsigned int width, const unsigned int height,
    float * const pointsHost)
{
  const unsigned int numInputPoints = width * height;
  const unsigned int pointsPerIteration = numInputPoints / 100;
  hipError_t status;

  // Copy the image to the GPU
  float *points;
  hipMalloc((void**)&points,
      sizeof(float) * numInputPoints * numInputDimensions);
  status = hipGetLastError();
  if (status != hipSuccess)
    throw std::runtime_error(std::string("CUDA Error: ") +
        hipGetErrorString(status));

  hipMemcpy(points, pointsHost,
      sizeof(float) * numInputPoints * numInputDimensions,
      hipMemcpyHostToDevice);
  status = hipGetLastError();
  if (status != hipSuccess)
    throw std::runtime_error(std::string("CUDA Error: ") +
        hipGetErrorString(status));

  // Pick a prime number close to 500 that is not a factor of the number of
  // pixels.  This is a step size to use for quasi-random training of the data
  unsigned int step = primes[3];
  for (unsigned int i = 0; i < 3; ++i)
  {
    if (numInputPoints % primes[i] != 0)
    {
      step = primes[i];
      break;
    }
  }

  dim3 gridDim(width, height, 1);

  // Train the network
  trainOnPoint<<<gridDim, networkSize>>>(points, step, pointsPerIteration);
  status = hipGetLastError();
  if (status != hipSuccess)
    throw std::runtime_error(std::string("CUDA Error: ") +
        hipGetErrorString(status));

  // Map the points to the output
  mapPoints<<<gridDim, networkSize>>>(points);
  status = hipGetLastError();
  if (status != hipSuccess)
    throw std::runtime_error(std::string("CUDA Error: ") +
        hipGetErrorString(status));

  // Copy the result back to the host
  hipMemcpy(pointsHost, points,
      sizeof(float) * numInputPoints * numInputDimensions,
      hipMemcpyDeviceToHost);
  status = hipGetLastError();
  if (status != hipSuccess)
    throw std::runtime_error(std::string("CUDA Error: ") +
        hipGetErrorString(status));

  hipFree(points);
  status = hipGetLastError();
  if (status != hipSuccess)
    throw std::runtime_error(std::string("CUDA Error: ") +
        hipGetErrorString(status));
}
