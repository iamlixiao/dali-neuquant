#include "hip/hip_runtime.h"
/*
 * Utilities.cpp
 *
 *  Created on: Oct 1, 2011
 *      Author: Dave
 */

#include "Utilities.h"

//------------------------------------------------------------------------------
// Name:        NodeDistance1::operator()
//------------------------------------------------------------------------------
float Utilities::NodeDistance1::operator ()(const unsigned int idx1,
    const unsigned int idx2)
{
  return static_cast<float>(idx1 < idx2 ? idx2 - idx1 : idx1 - idx2);
}

//------------------------------------------------------------------------------
// Name:        NodeDistance2::NodeDistance2()
//------------------------------------------------------------------------------
Utilities::NodeDistance2::NodeDistance2(const unsigned int cols) : cols(cols)
{ }

//------------------------------------------------------------------------------
// Name:        NodeDistance2::operator()
//------------------------------------------------------------------------------
float Utilities::NodeDistance2::operator()(const unsigned int idx1,
    const unsigned int idx2)
{
  const unsigned int row1 = idx1 / cols, col1 = idx1 % cols;
  const unsigned int row2 = idx2 / cols, col2 = idx2 % cols;

  return sqrt(static_cast<float>((row1 - row2) * (row1 - row2) +
      (col1 - col2) * (col1 - col2)));
}

//------------------------------------------------------------------------------
// Name:        NodeDistance3::NodeDistance3()
//------------------------------------------------------------------------------
Utilities::NodeDistance3::NodeDistance3(const unsigned int cols,
    const unsigned int sliceSize) : cols(cols), sliceSize(sliceSize)
{ }

//------------------------------------------------------------------------------
// Name:        NodeDistance3::operator()
//------------------------------------------------------------------------------
float Utilities::NodeDistance3::operator()(const unsigned int idx1,
    const unsigned int idx2)
{
  const unsigned int slice1 = idx1 / sliceSize,
      idx1InSlice = idx1 % sliceSize, row1 = idx1InSlice / cols,
      col1 = idx1InSlice % cols;
  const unsigned int slice2 = idx2 / sliceSize,
      idx2InSlice = idx2 % sliceSize, row2 = idx2InSlice / cols,
      col2 = idx2InSlice % cols;
  return sqrt(static_cast<float>((slice1 - slice2) * (slice1 - slice2) +
      (row1 - row2) * (row1 - row2) +
      (col1 - col2) * (col1 - col2)));
}

